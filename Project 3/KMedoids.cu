
#include <hip/hip_runtime.h>
//Filename: KMedoids.cu
//Author  : Saurabh A. Wani 


//Device kernel to calculate the distance of all points from 
// all possible medoids.
//Called with grid of 2-D blocks. 
//Result is stored in 2-D matrix. 
//First vector x	Contains all x coordinates of the data points.
//Second vector y	Contains all y coordinates of the data points.
//Third vecotr z	Contains all z coordinates of the data points.
//Output Matrix 	Stores the distances of all points from to their respective medoids.  
extern "C" __global__ void Kmed
	(double *x,
	 double *y,
	 double *z,
	 double **totDist,
	 int N)
	 {
	 	//First medoid.
	 	int row = blockIdx.y*blockDim.y + threadIdx.y;
	 	//Second medoid.
	 	int col = blockIdx.x*blockDim.x + threadIdx.x;
	 	double dist1;
	 	double dist2;
	 	// dist for given pair of medoids
	 		 	
	 	
	 	
	 	//Compute the distance between point and medoids. 
	 	if (row < N && col < N )
		 	{
		 		if (row == col)
			 		{
			 			totDist[row][col] = 0;
			 		}
		 		else
			 		{
			 			double tDist = 0.0;		
			 			for ( int k=0; k < N; k+=1)
			 				{
			 					dist1 = abs(x[row] - x[k]) + abs(y[row] - y[k]) + abs(z[row] - z[k]);
			 					dist2 = abs(x[col] - x[k]) + abs(y[col] - y[k]) + abs(z[col] - z[k]);
			 					if (dist1 < dist2)
			 						{
			 							tDist = tDist + dist1;
			 						}
			 					else
			 						{
			 							tDist = tDist + dist2;
			 						}
			 				}
			 			totDist[row][col] = tDist;
		 			}
		 	}					
	 						
	 						
	 						
	 						
}	 						
